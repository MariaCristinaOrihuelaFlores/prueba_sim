#include "hip/hip_runtime.h"
/*================================================
 *
 * CUDA MEX general beamformer for USTB
 *
 * Stefano Fiorentini <stefano.fiorentini@ntnu.no>
 * Last edit 01.02.2023
 *
 *================================================*/

 //Inputs
 //prhs[0]	 channel_data [time, channel, wave, frame]
 //prhs[1]   sampling frequency (Hz)
 //prhs[2]	 initial time (s)

 //prhs[3]	 transmit apodization [pixel, wave]
 //prhs[4]	 receive apodization [pixel, channel]

 //prhs[5]	 transmit delay [pixel, wave]
 //prhs[6]	 receive delay [pixel, channel]

 //prhs[7]   modulation frequency (Hz)
 //prhs[8]   sum mode 0 -> NONE, 1->RX, 2->TX, 3->BOTH
 //prhs[9]   gpu ID (default is 0)
         
 //Output
 //plhs[0]   beanformed data [pixel, channel, wave, frame]

#include <mex.h>
#include <matrix.h>

#include <math.h>
#include <stdbool.h>
#include <string.h>

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

// Constants
#define eps 1E-6f
#define pi acosf(-1.0)
#define thread_per_block 64

// Interpolation function
__device__  inline hipFloatComplex lerp(hipFloatComplex v0, hipFloatComplex v1, float t)
{
    hipFloatComplex v;
    
    v.x = fma(t, v1.x, fma(-t, v0.x, v0.x));
    v.y = fma(t, v1.y, fma(-t, v0.y, v0.y));

    return v;
}

// Beamforming kernel
__global__ void beamform(const int N_pixels, const int N_channels, const int N_waves, const float Fs, hipFloatComplex* bf_data, const hipTextureObject_t tex,
	const float* __restrict__ tx_delay, const float* __restrict__ rx_delay, const float* __restrict__ tx_apod, const float* __restrict__ rx_apod, const float i0, const float wd)
{
	int pixel_idx = blockIdx.x * blockDim.x + threadIdx.x; // pixel idx
	int pixel_stride = blockDim.x * gridDim.x;

	extern __shared__ float t[];

	float *tDelay = t;
	float *tApod = (float*)&tDelay[blockDim.x*N_waves];

    // Load tx delay and tx apodization matrices in shared memory because they are read multiple times
	for (int i = pixel_idx; i < N_pixels; i += pixel_stride)
	{
		for (int j = 0; j < N_waves; j++)
        {
			tDelay[threadIdx.x+j*blockDim.x] = tx_delay[i + j * N_pixels];
			tApod[threadIdx.x+j*blockDim.x] = tx_apod[i + j * N_pixels];
		}
	}

	__syncthreads();

	for (int i = pixel_idx; i < N_pixels; i += pixel_stride)
	{
		for (int g = 0; g < N_channels; g++)
		{
			const float rApod = rx_apod[i + g * N_pixels];

            if (rApod > 0.0f)
            {
				const float rDelay = rx_delay[i + g * N_pixels];

                for (int j = 0; j < N_waves; j++)
                {
					const float apod = rApod * tApod[threadIdx.x+j*blockDim.x];

                    if (apod > 0.0f)
                    {
						const float delay = rDelay + tDelay[threadIdx.x+j*blockDim.x];
                        const float denay = fma(delay, Fs, -i0);

                        hipFloatComplex phase;

                        __sincosf(wd * delay, &phase.y, &phase.x);

				        const float n = denay - floor(denay);

                        const hipFloatComplex val = lerp(tex2D<hipFloatComplex>(tex, denay, g + j * N_channels), 
                                                        tex2D<hipFloatComplex>(tex, denay+1.0f, g + j * N_channels), n);

                        bf_data[i].x = fma((val.x * phase.x - val.y * phase.y), apod, bf_data[i].x);
                        bf_data[i].y = fma((val.x * phase.y + val.y * phase.x), apod, bf_data[i].y);
                    }
                }
            }
        }
	}
}

#define cudaErrorCheck(arg) { cudaAssert((arg), __LINE__); }
inline void cudaAssert(hipError_t code, int line)
{
	if (code != hipSuccess)
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:GPU", "CUDA error: %s in line %d\n", hipGetErrorString(code), line);
	}
}

void mexCheckArguments(int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[]);

void mexFunction(int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[])
{

	// Check arguments
	mexCheckArguments(nlhs, plhs, nrhs, prhs);

	// Extract relevant parameters

	size_t* channel_size = (size_t*) mxGetDimensions(prhs[0]);
	size_t* tx_delay_size = (size_t*) mxGetDimensions(prhs[5]);

	size_t N_times	= channel_size[0];		// number of time samples
	size_t N_channels = channel_size[1];	// number of channels
	size_t N_waves = (mxGetNumberOfDimensions(prhs[0]) > 2) ? channel_size[2] : 1;	// number of waves
	size_t N_frames = (mxGetNumberOfDimensions(prhs[0]) > 3) ? channel_size[3] : 1;	// number of frames
	size_t N_pixels = tx_delay_size[0];		// number of pixels

	float Fs = *mxGetSingles(prhs[1]);		// Sampling frequency
	float t0 = *mxGetSingles(prhs[2]);		// Initial time
	float Fd = *mxGetSingles(prhs[7]);		// Modulation frequency
	float i0 = t0 * Fs;               // Normalised initial sample

	float wd = fabsf(Fd) > eps ? 2 * pi * Fd : 0.0;		// Demodulation frequency expressed in rad/s

	// Allocate beamformed data matrix in RAM
	size_t beamformed_size[4];
	beamformed_size[0] = N_pixels;  
	beamformed_size[1] = 1;			
	beamformed_size[2] = 1;			
	beamformed_size[3] = N_frames; 
	plhs[0] = mxCreateNumericArray(4, (const size_t*)&beamformed_size, mxSINGLE_CLASS, mxCOMPLEX);

    // Set gpuDevice to run CUDA code
  	int dev = *mxGetInt32s(prhs[9]);
    cudaErrorCheck(hipSetDevice(dev))
    
    // Get shared memory per block size of the selected GPU
    // int sharedMemPerBlock;
    //cudaErrorCheck(getCudaAttribute<int>(&sharedMemPerBlock,
    //                      hipDeviceAttributeMaxSharedMemoryPerBlock, dev));

	// Get pointer to beamformed data and pin memory for asynchronous memory transfer with the GPU
	mxComplexSingle* host_bf_data = mxGetComplexSingles(plhs[0]);
	cudaErrorCheck(hipHostRegister(host_bf_data, beamformed_size[0] * beamformed_size[1] * beamformed_size[2] * beamformed_size[3] * sizeof(mxComplexSingle), hipHostRegisterDefault)); // Pin paged memory for asynchronous transfers

	// Get pointer to channel data and pin memory for asynchronous memory transfer with the GPU
	mxComplexSingle* host_ch_data = mxGetComplexSingles(prhs[0]);
	cudaErrorCheck(hipHostRegister(host_ch_data, N_times * N_channels * N_waves * N_frames * sizeof(mxComplexSingle), hipHostRegisterDefault)); // Pin paged memory for asynchronous transfers

	// Transfer delay and apodization matrices to GPU
	// Retrieve pointer to host arrays
	float* host_tx_delay = mxGetSingles(prhs[5]);
	float* host_tx_apod = mxGetSingles(prhs[3]);
	float* host_rx_delay = mxGetSingles(prhs[6]);
	float* host_rx_apod = mxGetSingles(prhs[4]);

	// Allocate device memory
	float* device_tx_delay;
	float* device_tx_apod;
	float* device_rx_delay;
	float* device_rx_apod;

	cudaErrorCheck(hipMalloc((void**)&device_tx_delay, N_pixels * N_waves * sizeof(float)));
	cudaErrorCheck(hipMalloc((void**)&device_tx_apod, N_pixels * N_waves * sizeof(float)));
	cudaErrorCheck(hipMalloc((void**)&device_rx_delay, N_pixels * N_channels * sizeof(float)));
	cudaErrorCheck(hipMalloc((void**)&device_rx_apod, N_pixels * N_channels * sizeof(float)));

	// Transfer data
	cudaErrorCheck(hipMemcpy(device_tx_delay, host_tx_delay, N_pixels * N_waves * sizeof(float), hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(device_tx_apod, host_tx_apod, N_pixels * N_waves * sizeof(float), hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(device_rx_delay, host_rx_delay, N_pixels * N_channels * sizeof(float), hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpy(device_rx_apod, host_rx_apod, N_pixels * N_channels * sizeof(float), hipMemcpyHostToDevice));

	// If only one frame has to be processed, then only allocate one stream, otherwise allocate 2
	size_t N_streams = (N_frames > 1) ? 2 : 1;

	// Allocate device memory for beamformed data
	hipFloatComplex** device_bf_data = (hipFloatComplex**) malloc(N_streams * sizeof(hipArray**));
	
	for (size_t n_stream = 0; n_stream < N_streams; n_stream++)
	{
		cudaErrorCheck(hipMalloc((void**)&device_bf_data[n_stream], N_pixels * sizeof(hipFloatComplex)));
	}

	// Allocate an array of 2D hipArray and a cudaTextureObjects
	// Need 2 elements in the array to allow for asynchronous operations
	hipArray** device_ch_data = (hipArray**)malloc(N_streams * sizeof(hipArray*)); // Array of pointers to cudaArrays
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat); // channel descriptor for a hipFloatComplex type.
	hipTextureObject_t* tex = (hipTextureObject_t*)malloc(N_streams * sizeof(hipTextureObject_t));

	for (size_t n_stream = 0; n_stream < N_streams; n_stream++)
	{
		cudaErrorCheck(hipMallocArray(&device_ch_data[n_stream], &channelDesc, N_times, N_channels*N_waves, hipArrayDefault)); // Allocate 2D texture

		// Input data properties
		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(hipResourceDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = device_ch_data[n_stream];

		// Texture properties
		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(hipTextureDesc));
		texDesc.filterMode = hipFilterModePoint; // nearest neighbour interpolation
		texDesc.normalizedCoords = false; // coordinates are not normalized [0, 1, ..., N_times-1]
		texDesc.addressMode[0] = hipAddressModeBorder; // out of bound coordinates are 0
        texDesc.addressMode[1] = hipAddressModeBorder; // out of bound coordinates are 0
		texDesc.readMode = hipReadModeElementType;

		// Texture Object
		cudaErrorCheck(hipCreateTextureObject(&tex[n_stream], &resDesc, &texDesc, NULL));
	}

	// Define block_size and N_blocks
	dim3 dimBlock = dim3(thread_per_block, 1, 1);
	dim3 dimGrid = dim3((N_pixels + dimBlock.x - 1) / dimBlock.x, 1, 1);

	// Setupt cudaStream for asynchronous operations
	hipStream_t* frame_stream = (hipStream_t*)malloc(N_streams * sizeof(hipStream_t));
	for (size_t n_stream = 0; n_stream < N_streams; n_stream++)
	{
		cudaErrorCheck(hipStreamCreate(&frame_stream[n_stream]));
	}

	// Beamforming loop
	for (size_t n_frame = 0; n_frame < N_frames; n_frame += N_streams)
	{
		size_t Nc_streams = (N_frames - n_frame) < N_streams |  N_streams == 1 ? 1 : N_streams;

		for (size_t n_stream = 0; n_stream < Nc_streams; n_stream++)
		{
			// Copy channel data into hipArray
			cudaErrorCheck(hipMemcpy2DToArrayAsync(device_ch_data[n_stream], 
            0, 0, &host_ch_data[(n_frame + n_stream) * N_waves * N_channels * N_times], 
            N_times * sizeof(hipFloatComplex), N_times * sizeof(hipFloatComplex), N_channels * N_waves, 
            hipMemcpyHostToDevice, frame_stream[n_stream]));

		}

		for (size_t n_stream = 0; n_stream < Nc_streams; n_stream++)
		{
			// Set device beamformed data to 0
			cudaErrorCheck(hipMemsetAsync(device_bf_data[n_stream], 0, N_pixels * sizeof(hipFloatComplex), frame_stream[n_stream]));

			// Call beamforming kernel
			beamform <<< dimGrid, dimBlock, 2*thread_per_block*N_waves*sizeof(float), frame_stream[n_stream] >>> ((int) N_pixels, (int) N_channels, (int) N_waves, Fs, device_bf_data[n_stream], tex[n_stream], device_tx_delay,
				device_rx_delay, device_tx_apod, device_rx_apod, i0, wd);
			cudaErrorCheck(hipPeekAtLastError());
		}

		for (size_t n_stream = 0; n_stream < Nc_streams; n_stream++)
		{
			// Transfer beamformed data back to host
			cudaErrorCheck(hipMemcpyAsync(&host_bf_data[(n_frame + n_stream) * N_pixels], device_bf_data[n_stream], N_pixels * sizeof(hipFloatComplex), hipMemcpyDeviceToHost, frame_stream[n_stream]));
		}
	} // end of frame loop

	for (size_t n_stream = 0; n_stream < N_streams; n_stream++)
	{
		// Destroy cudaStreams
		cudaErrorCheck(hipStreamDestroy(frame_stream[n_stream]));

		// Free Texture memory
		cudaErrorCheck(hipFreeArray(device_ch_data[n_stream]));
		cudaErrorCheck(hipDestroyTextureObject(tex[n_stream]));

		// Free beamformed data memory
		cudaErrorCheck(hipFree(device_bf_data[n_stream]));
	}

	cudaErrorCheck(hipFree(device_tx_apod));
	cudaErrorCheck(hipFree(device_tx_delay));
	cudaErrorCheck(hipFree(device_rx_apod));
	cudaErrorCheck(hipFree(device_rx_delay));

	// Unpin host memory
	cudaErrorCheck(hipHostUnregister(host_ch_data));
	cudaErrorCheck(hipHostUnregister(host_bf_data));
}

void mexCheckArguments(int nlhs, mxArray* plhs[], int nrhs, const mxArray* prhs[])
{
	// Check number of arguments
	if (nrhs != 10)
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:nrhs", "Wrong number of input arguments");
	}
	if (nlhs > 1)
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:nlhs", "Too many output arguments");
	}

	// Check that bmf.dimension is set to dimension.both
	if (*mxGetInt32s(prhs[8]) != 3)
	{
		mexErrMsgTxt("In this implementation only dimension.both is supported");
	}

	// Channel data
	// Check dimension
	if (mxGetNumberOfDimensions(prhs[0]) < 2 || mxGetNumberOfDimensions(prhs[0]) > 4)
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:Dimensions", "Wrong channel data format. Must be in the form [time, channel, wave, frame]");
	}
	// Get channel data size
	size_t* channel_size = (size_t*)mxGetDimensions(prhs[0]);

	// Check that channel data is of type complex float
	if (mxIsDouble(prhs[0]) && !mxIsComplex(prhs[0]))
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:Float", "Channel data must be complex float");
	}

	// Transmit delay
	// check dimensions
	if (mxGetNumberOfDimensions(prhs[5]) > 2)
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:Dimensions", "Wrong transmit delay matrix format. Expected 2 dimensions: [pixel, wave]");
	}
	// Get transmit delay matrix size
	size_t* tx_delay_size = (size_t*)mxGetDimensions(prhs[5]);

	// Check that the number of dimensions match
	if (mxGetNumberOfDimensions(prhs[5]) > 1 && tx_delay_size[1] != channel_size[2])
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:Dimensions", "Channel data and transmit delay size do not match");
	}

	// check that tx delay matrix is of type float
	if (mxIsDouble(prhs[5]))
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:Float", "The transmit delay must be of type float");
	}

	// Receive delay
	// check dimensions
	if (mxGetNumberOfDimensions(prhs[6]) > 2)
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:Dimensions", "Wrong receive delay matrix format. Expected 2 dimensions: [pixel, channels]");
	}
	// Get receive delay matrix size
	size_t* rx_delay_size = (size_t*)mxGetDimensions(prhs[6]);

	// Check that the number of dimensions match
	if (mxGetNumberOfDimensions(prhs[6]) > 1 && rx_delay_size[1] != channel_size[1])
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:Dimensions", "Channel data and receive delay size do not match");
	}

	// check that receive delay matrix is of type float
	if (mxIsDouble(prhs[6]))
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:Float", "The receive delay must be of type float");
	}

	// Transmit apodization
	// check dimensions
	if (mxGetNumberOfDimensions(prhs[3]) > 2)
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:Dimensions", "Wrong transmit apodization matrix format. Expected 2 dimensions: [pixel, waves]");
	}
	// Get tx data size
	size_t* tx_apodization_size = (size_t*)mxGetDimensions(prhs[3]);

	// Check that the number of dimensions match
	if (mxGetNumberOfDimensions(prhs[3]) > 1 && tx_apodization_size[1] != channel_size[2])
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:Dimensions", "Channel data and transmit apodization size do not match");
	}

	// check that tx apodization matrix is of type float
	if (mxIsDouble(prhs[3]))
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:Float", "The receive delay must be of type float");
	}

	// Receive apodization
	// check dimensions
	if (mxGetNumberOfDimensions(prhs[4]) > 2)
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:Dimensions", "Wrong receive apodization matrix format. Expected 2 dimensions: [pixel, channels]");
	}
	// Get receive delay matrix size
	size_t* rx_apodization_size = (size_t*)mxGetDimensions(prhs[4]);

	// Check that the number of dimensions match
	if (mxGetNumberOfDimensions(prhs[4]) > 1 && rx_apodization_size[1] != channel_size[1])
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:Dimensions", "Channel data and receive apodization size do not match");
	}

	// check that receive delay matrix is of type float
	if (mxIsDouble(prhs[4]))
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:Float", "The receive apodization must be of type float");
	}

	// Sampling frequency
	// check dimensions
	if (!mxIsScalar(prhs[1]))
	{
		mexErrMsgTxt("The sampling frequency should be a scalar");
	}
	// check single precision
	if (mxIsDouble(prhs[1]))
	{
		mexErrMsgTxt("The sampling frequency should be of type float");
	}

	// Initial time
	// check dimensions
	if (!mxIsScalar(prhs[2]))
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:Float", "The the initial time must be a scalar");
	}
	// check single precision
	if (mxIsDouble(prhs[2]))
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:Float", "The initial time must be of type float");
	}

	// Modulation frequency
	// check dimension
	if (!mxIsScalar(prhs[7]))
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:Scalar", "The modulation frequency must be a scalar");
	}
	// check single precision
	if (mxIsDouble(prhs[7]))
	{
		mexErrMsgIdAndTxt("Toolbox:SRP_SRC:Float", "The modulation frequency must be of type float");
	}
}